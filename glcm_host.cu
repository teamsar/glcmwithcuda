#include "hip/hip_runtime.h"

   #include <hip/hip_runtime.h>
   #include <hip/hip_runtime.h>
   #include <stdio.h>
   #include <stdlib.h>
   #include <time.h>
   #include "kuda.h"
   #include "lodepng.h"
// Banyak nx * nx Matrix
// Banyak Max * Max Matrix
int Max;


void printMatrixGlcm(int *C, const int Max,int degree,int gambar)
{
    int *ic = C;
    FILE * fp = NULL;
    if(gambar==128){
        if(degree==0){
            fp = fopen("./data/GLCM/Sample128/Sudut_0/matrix_glcm_0_host.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/GLCM/Sample128/Sudut_90/matrix_glcm_90_host.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/GLCM/Sample128/Sudut_180/matrix_glcm_180_host.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/GLCM/Sample128/Sudut_270/matrix_glcm_270_host.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/GLCM/Sample128/Sudut_45/matrix_glcm_45_host.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/GLCM/Sample128/Sudut_135/matrix_glcm_135_host.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/GLCM/Sample128/Sudut_225/matrix_glcm_225_host.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/GLCM/Sample128/Sudut_315/matrix_glcm_315_host.txt", "w+");
        }
    }
    else if(gambar==256){
        if(degree==0){
            fp = fopen("./data/GLCM/Sample256/Sudut_0/matrix_glcm_0_host.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/GLCM/Sample256/Sudut_90/matrix_glcm_90_host.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/GLCM/Sample256/Sudut_180/matrix_glcm_180_host.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/GLCM/Sample256/Sudut_270/matrix_glcm_270_host.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/GLCM/Sample256/Sudut_45/matrix_glcm_45_host.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/GLCM/Sample256/Sudut_135/matrix_glcm_135_host.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/GLCM/Sample256/Sudut_225/matrix_glcm_225_host.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/GLCM/Sample256/Sudut_315/matrix_glcm_315_host.txt", "w+");
        }
    }
    else if(gambar==512){
        if(degree==0){
            fp = fopen("./data/GLCM/Sample512/Sudut_0/matrix_glcm_0_host.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/GLCM/Sample512/Sudut_90/matrix_glcm_90_host.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/GLCM/Sample512/Sudut_180/matrix_glcm_180_host.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/GLCM/Sample512/Sudut_270/matrix_glcm_270_host.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/GLCM/Sample512/Sudut_45/matrix_glcm_45_host.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/GLCM/Sample512/Sudut_135/matrix_glcm_135_host.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/GLCM/Sample512/Sudut_225/matrix_glcm_225_host.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/GLCM/Sample512/Sudut_315/matrix_glcm_315_host.txt", "w+");
        }
    }
    else if(gambar==1024){
        if(degree==0){
            fp = fopen("./data/GLCM/Sample1024/Sudut_0/matrix_glcm_0_host.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/GLCM/Sample1024/Sudut_90/matrix_glcm_90_host.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/GLCM/Sample1024/Sudut_180/matrix_glcm_180_host.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/GLCM/Sample1024/Sudut_270/matrix_glcm_270_host.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/GLCM/Sample1024/Sudut_45/matrix_glcm_45_host.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/GLCM/Sample1024/Sudut_135/matrix_glcm_135_host.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/GLCM/Sample1024/Sudut_225/matrix_glcm_225_host.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/GLCM/Sample1024/Sudut_315/matrix_glcm_315_host.txt", "w+");
        }
    }


    if(fp == NULL){
        printf("Error creating results file\n");
        exit(1);
    }
    for (int iy = 0; iy <Max; iy++)
    {
        for (int ix = 0; ix <Max; ix++)
        {
            fprintf(fp, "%d  ", ic[ix]);

        }
        fprintf(fp, "\n\n");
        ic += (Max);

    }

    printf("\n");
    fclose(fp);
    return;
}

void printMatrixnxormalization(float *C, const int Max,int degree,int gambar)
{
    float *ic = C;
    FILE * fp = NULL;
    if(gambar==128){
        if(degree==0){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_0/matrix_normalisasi_0_host.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_90/matrix_normalisasi_90_host.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_180/matrix_normalisasi_180_host.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_270/matrix_normalisasi_270_host.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_45/matrix_normalisasi_45_host.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_135/matrix_normalisasi_135_host.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_225/matrix_normalisasi_225_host.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_315/matrix_normalisasi_315_host.txt", "w+");
        }
    }
    else if(gambar==256){
        if(degree==0){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_0/matrix_normalisasi_0_host.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_90/matrix_normalisasi_90_host.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_180/matrix_normalisasi_180_host.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_270/matrix_normalisasi_270_host.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_45/matrix_normalisasi_45_host.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_135/matrix_normalisasi_135_host.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_225/matrix_normalisasi_225_host.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_315/matrix_normalisasi_315_host.txt", "w+");
        }
    }
    else if(gambar==512){
        if(degree==0){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_0/matrix_normalisasi_0_host.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_90/matrix_normalisasi_90_host.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_180/matrix_normalisasi_180_host.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_270/matrix_normalisasi_270_host.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_45/matrix_normalisasi_45_host.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_135/matrix_normalisasi_135_host.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_225/matrix_normalisasi_225_host.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_315/matrix_normalisasi_315_host.txt", "w+");
        }
    }
    else if(gambar==1024){
        if(degree==0){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_0/matrix_normalisasi_0_host.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_90/matrix_normalisasi_90_host.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_180/matrix_normalisasi_180_host.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_270/matrix_normalisasi_270_host.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_45/matrix_normalisasi_45_host.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_135/matrix_normalisasi_135_host.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_225/matrix_normalisasi_225_host.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_315/matrix_normalisasi_315_host.txt", "w+");
        }
    }
    if(fp == NULL){
        printf("Error creating results file\n");
        exit(1);
    }
    for (int iy = 0; iy < Max; iy++)
    {
        for (int ix = 0; ix <Max; ix++)
        {

            fprintf(fp, "%.7f  ", ic[ix]);

        }
        fprintf(fp, "\n\n");
        ic +=Max;

    }

    printf("\n");
    fclose(fp);
    return;
}


// void calculate_glcm_host(int *matrix,int *glcm,int nx,int ny,int Max){
//     int i,j;
//     for(i=0;i<nx;i++){
//         for(j=0;j<ny;j++){
//             glcm[matrix[i]][matrix[j]] +=1;
//         }
//     }
// }

//calculate glcm



void calculate_glcm_host(int *matrix,int *glcm,int N,int Max){
    int i,j,k,l;

    for(i=0;i<N;i++){
        for(j=0;j<N-1;j++){
            for(k=0;k<Max;k++){
                for(l=0;l<Max;l++){
                    if(matrix[Max*i+j]==k&&matrix[Max*i+(j+1)]==l){
                        //printf("%d,%d,%d,%d\n",matrix[Max*i+j],k,matrix[Max*i+(j+1)],l);
                        glcm[Max*k+l] +=1;
                    }
                }
            }
        }
    }
}


__global__ void blok1_0(int *matrix,int *glcm,int N, int Max){
    int i,j,k,l;

    for(i=0;i<N;i++){
        for(j=0;j<N-1;j++){
            for(k=0;k<Max;k++){
                for(l=0;l<Max;l++){
                    if(matrix[N*i+j]==k && matrix[N*i+(j+1)]==l){
                        //printf("%d,%d,%d,%d\n",matrix[Max*i+j],k,matrix[Max*i+(j+1)],l);
                        atomicAdd(&glcm[Max*k+l],1);
                    }
                }
            }
        }
    }

}


  __global__ void blok1_45(int *matrix,int *glcm,int N, int Max){
      int i,j,k,l;

      for(i=1;i<N;i++){
          for(j=0;j<N-1;j++){
            for(k=0;k<Max;k++){
                for(l=0;l<Max;l++){
                    if(matrix[N*i+j]==k && matrix[N*(i-1)+(j+1)]==l){
                        //printf("%d,%d,%d,%d\n",matrix[Max*i+j],k,matrix[Max*i+(j+1)],l);
                        atomicAdd(&glcm[Max*matrix[N*i+j]+matrix[N*(i-1)+(j+1)]] ,1);
                    }
                }
            }
             

          }
      }
  }

  __global__ void blok1_90(int *matrix,int *glcm,int N, int Max){
    int i,j,k,l;

    for(i=1;i<N;i++){
        for(j=0;j<N;j++){
            for(k=0;k<Max;k++){
                for(l=0;l<Max;l++){
                    if(matrix[N*i+j]==k &&matrix[N*(i-1)+j]==l){
                        //printf("%d,%d,%d,%d\n",matrix[Max*i+j],k,matrix[Max*i+(j+1)],l);
                        atomicAdd(&glcm[Max*matrix[N*i+j]+matrix[N*(i-1)+j]] ,1);
                    }                    
                }
            }  
          

        }
    }
}


__global__ void blok1_135(int *matrix,int *glcm,int N, int Max){
    int i,j,k,l;

    for(i=1;i<N;i++){
        for(j=1;j<N;j++){
            for(k=0;k<Max;k++){
                for(l=0;l<Max;l++){
                    if(matrix[N*i+j]==k &&matrix[N*(i-1)+(j-1)]==l){
                        //printf("%d,%d,%d,%d\n",matrix[Max*i+j],k,matrix[Max*i+(j+1)],l);
                        atomicAdd(&glcm[Max*matrix[N*i+j]+matrix[N*(i-1)+(j-1)]] ,1);
                    }                    
                }
            }

        }
    }
}

__global__ void blok1_180(int *matrix,int *glcm,int N, int Max){
    int i,j,k,l;

    for(i=0;i<N;i++){
        for(j=1;j<N;j++){
            for(k=0;k<Max;k++){
                for(l=0;l<Max;l++){
                    if(matrix[N*i+j]==k &&matrix[N*i+(j-1)]==l){
                        //printf("%d,%d,%d,%d\n",matrix[Max*i+j],k,matrix[Max*i+(j+1)],l);
                        atomicAdd(&glcm[Max*matrix[N*i+j]+matrix[N*i+(j-1)]] ,1);
                    }                    
                }
            }
           

        }
    }
}

__global__ void blok1_225(int *matrix,int *glcm,int N, int Max){
    int i,j,k,l;

    for(i=0;i<N-1;i++){
        for(j=1;j<N;j++){
            for(k=0;k<Max;k++){
                for(l=0;l<Max;l++){
                    if(matrix[N*i+j]==k &&matrix[N*(i+1)+(j-1)]==l){
                        //printf("%d,%d,%d,%d\n",matrix[Max*i+j],k,matrix[Max*i+(j+1)],l);
                        atomicAdd(&glcm[Max*matrix[N*i+j]+matrix[N*(i+1)+(j-1)]] ,1);
                    }                    
                }
            }
           

        }
    }
}


__global__ void blok1_270(int *matrix,int *glcm,int N, int Max){
    int i,j,k,l;

    for(i=0;i<N-1;i++){
        for(j=0;j<N;j++){
            for(k=0;k<Max;k++){
                for(l=0;l<Max;l++){
                    if(matrix[N*i+j]==k &&matrix[N*(i+1)+j]==l){
                        //printf("%d,%d,%d,%d\n",matrix[Max*i+j],k,matrix[Max*i+(j+1)],l);
                        atomicAdd(&glcm[Max*matrix[N*i+j]+matrix[N*(i+1)+j]] ,1);
                    }                    
                }
            }
           

        }
    }
}

__global__ void blok1_315(int *matrix,int *glcm,int N, int Max){
    int i,j,k,l;

    for(i=0;i<N-1;i++){
        for(j=0;j<N-1;j++){
            for(k=0;k<Max;k++){
                for(l=0;l<Max;l++){
                    if(matrix[N*i+j]==k &&matrix[N*(i+1)+(j+1)]==l){
                        //printf("%d,%d,%d,%d\n",matrix[Max*i+j],k,matrix[Max*i+(j+1)],l);
                        atomicAdd(&glcm[Max*matrix[N*i+j]+matrix[N*(i+1)+(j+1)]] ,1);
                    }                    
                }
            }
           

        }
    }
}

// __global__ void blok1_45(int *matrix,int *glcm,int N, int Max){
//     int i,j,k,l;

//     for(i=1;i<N;i+2){
//         for(j=1;j<N;j++){
//             for(k=0;k<Max;k++){
//                 for(l=0;l<Max;l++){
//                     if(matrix[N*i+j]==k&&matrix[N*i+(j+1)]==l){
//                         //printf("%d,%d,%d,%d\n",matrix[Max*i+j],k,matrix[Max*i+(j+1)],l);
//                         atomicAdd(&glcm[Max*k+l] ,1);
//                     }
//                 }
//             }
//         }
//     }

// }

__global__ void Mul(float *glcm,float *mulMatrix,int Max,float *sumMatrix){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // int Index = iy * nx + ix;

    for (int k = 0; k < Max; k++) {
        // Accumulate results for a single element
        // c[row * nx + col] += a[row * nx + k] * b[k * nx + col];
        // printf("C[%d] = a[%d] * b[%d]\n",row * nx + col,row * nx + k, k * nx + col);
        atomicAdd(&mulMatrix[row * Max + col],glcm[row * Max + k] * glcm[k * Max + col]);
        // atomicAdd(&sumMatrix[0],mulMatrix[row * Max + col]);
    }
}


__global__ void Jumlah(float *sumMatrix,float *mulMatrix){
    int Index = blockIdx.x * blockDim.x + threadIdx.x;
    // if(Index<1) printf("%f",mulMatrix[0]);
    atomicAdd(&sumMatrix[0],mulMatrix[Index]);

}

__global__ void AddToitTranspose(int *transposed,int *glcm,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;


    transposed[row*Max+col]=glcm[row*Max+col]+glcm[col*Max+row];

}

__global__ void normalization(int *glcm,float *norm,int Max,int sum){
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * Max + ix;
    __syncthreads();
    if(idx<(Max+1)*(Max+1)){
        norm[idx]=float(glcm[idx])/float(sum);
    }
}
__global__ void calculate_contrast(float *norm,float *contrast,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        atomicAdd(&contrast[0],((row-col)*(row-col))*norm[row*Max+col]);
    }

}

__global__ void calculate_IDM(float *norm,float *IDM,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        atomicAdd(&IDM[0],norm[row*Max+col] / (1+((row-col)*(row-col))) );
    }

}

__global__ void calculate_entropy(float *norm,float *entropy,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        atomicAdd(&entropy[0],(norm[row*Max+col] * log10f(norm[row*Max+col])) );

    }

}


__global__ void calculate_ASM(float *norm,float *ASM,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
       // printf("nilai %d %d %d %f\n",row*Max+col,row,col,norm[row*Max+col]);
        atomicAdd(&ASM[0],norm[row*Max+col]*norm[row*Max+col]);
        //printf("%f\n",ASM[0]);
    }
}

__global__ void calculate_miu_i(float *norm,float *miu_i,int Max){
    //printf("%d\n",max);
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        //printf("nilai %d %d %d %f\n",row*Max+col,row,col,norm[row*Max+col]);
        atomicAdd(&miu_i[0],row*norm[row*Max+col]);
    }
}



__global__ void calculate_miu_j(float *norm,float *miu_j,int Max){
    //printf("%d\n",max);
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        //printf("nilai %d %d %d %f\n",row*Max+col,row,col,norm[row*Max+col]);
        atomicAdd(&miu_j[0],col*norm[row*Max+col]);
    }
}

__global__ void calculate_std_i(float *norm,float *std_i,float*miu_i,int Max){
    //printf("%d\n",max);
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        //printf("nilai %d %d %d %f\n",row*Max+col,row,col,norm[row*Max+col]);
        atomicAdd(&std_i[0],norm[row*Max+col] * ((row-miu_i[0])*(row-miu_i[0])));
    }
}


__global__ void calculate_std_j(float *norm,float *std_i,float *miu_j,int Max){
    //printf("%d\n",max);
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        //printf("nilai %d %d %d %f\n",row*Max+col,row,col,norm[row*Max+col]);
        atomicAdd(&std_i[0],norm[row*Max+col]*(((col-miu_j[0])*(col-miu_j[0]))));
    }

}__global__ void calculate_korelasi(float *norm,float *korelasi,float *miu_i,float *std_i,float *miu_j,float *std_j,int Max){
    //printf("%d\n",max);
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        //printf("nilai %d %d %d %f\n",row*Max+col,row,col,norm[row*Max+col]);
        atomicAdd(&korelasi[0],(((row-miu_i[0])*(col-miu_j[0]))*norm[row*Max+col])/(std_i[0]*std_j[0]));

    }
}

__global__ void calculate_variance(float *norm,float *variance,float *miu_i,int Max){
    //printf("%d\n",max);
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        //printf("nilai %d %d %d %f\n",row*Max+col,row,col,norm[row*Max+col]);
        atomicAdd(&variance[0],((row-miu_i[0])*(row-miu_i[0]))*norm[row*Max+col]);

    }
}

__global__ void calculate_sumaverage(float *norm,float *sav,int Max){
    //printf("%d\n",max);
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int k;
    for(k=2;k<2*Max;k++){
        if((row+col)==k){
            atomicAdd(&sav[0],k*(1*norm[row*Max+col]));
        }
        else{
            atomicAdd(&sav[0],0);
        }
    }
}

__global__ void calculate_sumentropy(float *norm,float *sen,int Max){
    //printf("%d\n",max);
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int k;
    for(k=2;k<2*Max;k++){
        if((row+col)==k && norm[row*Max+col]>0){
            //printf("%f\n",norm[row*Max+col]);
            atomicAdd(&sen[0],(1*norm[row*Max+col])*(log10(1*norm[row*Max+col])));
        }
        else{
            atomicAdd(&sen[0],0);
        }
    }
}

__global__ void calculate_sumvariance(float *norm,float *sva,float *sen,int Max){
    //printf("%d\n",max);
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int k;
    for(k=2;k<2*Max;k++){
        if((row+col)==k && norm[row*Max+col]>0){
            //printf("%f\n",norm[row*Max+col]);
            atomicAdd(&sva[0],((k-sen[0])*(k-sen[0]))*(1*norm[row*Max+col]));
        }
        else{
            atomicAdd(&sva[0],0);
        }
    }
}

__global__ void calculate_differenceentropy(float *norm,float *den,int Max){
    //printf("%d\n",max);
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int k;
    for(k=0;k<Max-1;k++){
        if(abs(row-col)==k && norm[row*Max+col]>0){
            //printf("%f\n",norm[row*Max+col]);
            atomicAdd(&den[0],(1*norm[row*Max+col])*(log10(1*norm[row*Max+col])));
        }
        else{
            atomicAdd(&den[0],0);
        }
    }
}

__global__ void calculate_HX(float *norm,float *HX,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        atomicAdd(&HX[0],norm[row*Max+col]*log10f(norm[row*Max+col]));
    }
}


__global__ void calculate_HY(float *norm,float *HY,int Max){
    //printf("%d\n",max);
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(norm[row*Max+col]>0){
        atomicAdd(&HY[0],norm[row*Max+col]*log10f(norm[row*Max+col]));

    }
}

__global__ void calculate_HXY1(float *norm,float *HXY1,int Max){
    //printf("%d\n",max);
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        //printf("%.13f %.13f %f %f \n",norm[row],norm[col],norm[row*Max+col],log10f(norm[row]*norm[col]));
        atomicAdd(&HXY1[0],norm[row*Max+col]*log10f(norm[row*Max+col]));
    }
}

__global__ void calculate_dva(float *norm,float *dva,int Max){
    //printf("%d\n",max);
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int k;
    for(k=0;k<Max-1;k++){
        if(abs(row-col)==k && norm[row*Max+col]>0){
            //printf("%f\n",norm[row*Max+col]);
            atomicAdd(&dva[0],(k*k)*(1*norm[row*Max+col]));
        }
        else{
            atomicAdd(&dva[0],0);
        }
    }
}

void takeimagevalue(const char* filename, rgb_image *img)
{

     unsigned error;
     unsigned char* png;
     size_t pngsize;;

     lodepng_load_file(&png, &pngsize, filename);
     error = lodepng_decode32(&img->image, &img->width, &img->height, png, pngsize);

     if(error) printf("error %u: %s\n", error, lodepng_error_text(error));

}

void transformToGrayCuda(rgb_image *img){
	unsigned char* image = img->image;
    unsigned char* image_d;
    unsigned int  width = img->width;
    unsigned int height = img->height;
    int n =width*height;
    size_t size = n * 4 * sizeof(unsigned char);


	int device_count = 0;
	hipError_t status = hipGetDeviceCount(&device_count);

	status = hipMalloc((void **) &image_d, size);


	hipMemcpy(image_d, image,  size, hipMemcpyHostToDevice);

	dim3 block_size(16, 16);
	dim3 num_blocks(img->width / block_size.x, img->height / block_size.y);
    setPixelToGrayscale<<<num_blocks, block_size>>>(image_d, img->width, img->height);



	hipMemcpy(image, image_d, size, hipMemcpyDeviceToHost);

	hipFree(image_d);
}


__global__
void setPixelToGrayscale(unsigned char *image, unsigned width, unsigned height)
{
    float gray;
    float r, g, b;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		r = image[4 * width * y + 4 * x + 0];
		g = image[4 * width * y + 4 * x + 1];
		b = image[4 * width * y + 4 * x + 2];
		gray =.299f*r + .587f*g + .114f*b;
		image[4 * width * y + 4 * x + 0] = gray;
		image[4 * width * y + 4 * x + 1] = gray;
		image[4 * width * y + 4 * x + 2] = gray;
		image[4 * width * y + 4 * x + 3] = 255;
	}

}

void saveimagegray(const char* filename, rgb_image *img)
{
  /*Encode the image*/
  unsigned error = lodepng_encode32_file(filename, img->image, img->width, img->height);

  /*if there's an error, display it*/
  if(error) printf("error %u: %s\n", error, lodepng_error_text(error));
}

int main(int argc, char *argv[]){


    char *d;
    long deg =strtol(argv[2],&d,10);
    int degree=deg;

    const char* filename = argc > 1 ? argv[1] : "test.png";
    rgb_image img;
    takeimagevalue(filename, &img);
    transformToGrayCuda(&img);
    int nx =img.width;
    int ny =img.height;
    printf("%s %d degre Starting...\n", argv[0],degree,nx);
    printf("%d %d\n",nx,ny);
    int *matrix,*glcm,*transposed;
    float *norm,*mulMatrix,*sumMatrix;
    float*ASM,*contrast,*IDM,*entropy,*miu_i,*miu_j,*std_i,*std_j,*korelasi,*variance,*sav,*sen,*sva,*den,*HX,*HY,*HXY1,*dva;
    hipMallocManaged(&matrix, (nx * ny) * sizeof(int));

    for(int i = 0 ; i < (nx * nx) ; ++i){
        matrix[i] = img.image[i];
        if(matrix[i] > Max){
            Max = matrix[i];
        }
    }

    // for(int i = 0 ; i < nx ; ++i){
    //     for(int j = 0 ; j < nx ; ++j){
    //         printf("%4d",matrix[i * nx + j]);
    //     }
    //     printf("\n");
    // }
    //printf("\n\n");
    Max = Max + 1; // karena index dimulai dari 0 dan Maximum 3 ( 0 - 3 = 4 ) jadi Max ditambah 1;

    hipMallocManaged(&glcm, (Max * Max) * sizeof(int));
    hipMallocManaged(&transposed, (Max * Max) * sizeof(int));
    hipMallocManaged(&mulMatrix, (Max * Max) * sizeof(float));
    hipMallocManaged(&sumMatrix, (Max * Max) * sizeof(float));
    hipMallocManaged(&norm, (Max * Max) * sizeof(float));
    for(int i = 0 ; i < (Max * Max) ; ++i){
        glcm[i] = 0;
        transposed[i] = 0;
        mulMatrix[i] = 0;
    }



    hipMallocManaged(&ASM, (Max * Max) * sizeof(float));
    hipMallocManaged(&contrast, (Max * Max) * sizeof(float));
    hipMallocManaged(&IDM, (Max * Max) * sizeof(float));
    hipMallocManaged(&entropy, (Max * Max) * sizeof(float));
    hipMallocManaged(&miu_i, (Max * Max) * sizeof(float));
    hipMallocManaged(&miu_j, (Max * Max) * sizeof(float));
    hipMallocManaged(&std_i, (Max * Max) * sizeof(float));
    hipMallocManaged(&std_j, (Max * Max) * sizeof(float));
    hipMallocManaged(&korelasi, (Max * Max) * sizeof(float));
    hipMallocManaged(&variance, (Max * Max) * sizeof(float));
    hipMallocManaged(&sav, (Max * Max) * sizeof(float));
    hipMallocManaged(&sen, (Max * Max) * sizeof(float));
    hipMallocManaged(&sva, (Max * Max) * sizeof(float));
    hipMallocManaged(&den, (Max * Max) * sizeof(float));
    hipMallocManaged(&HX, (Max * Max) * sizeof(float));
    hipMallocManaged(&HY, (Max * Max) * sizeof(float));
    hipMallocManaged(&dva, (Max * Max) * sizeof(float));
    hipMallocManaged(&HXY1, (Max * Max) * sizeof(float));

    dim3 block(2 ,2);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    dim3 grids((Max + block.x - 1) / block.x, (Max + block.y - 1) / block.y);
    clock_t start, end;
    double t = 0;
    start = clock();
    // invoke kernel for calculation
    if(degree==0){
        blok1_0<<<1,1>>>(matrix,glcm,nx,Max);
        hipDeviceSynchronize();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    else if(degree ==180){
        blok1_180<<<1,1>>>(matrix,glcm,nx,Max);;
        hipDeviceSynchronize();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    else if(degree==270){
        blok1_270<<<1,1>>>(matrix,glcm,nx,Max);;
        hipDeviceSynchronize();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    else if(degree==90){
        blok1_90<<<1,1>>>(matrix,glcm,nx,Max);;
        hipDeviceSynchronize();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    else if(degree==45){
        blok1_45<<<1,1>>>(matrix,glcm,nx,Max);;
        hipDeviceSynchronize();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    else if(degree==135){
        blok1_135<<<1,1>>>(matrix,glcm,nx,Max);;
        hipDeviceSynchronize();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    else if(degree==225){
        blok1_225<<<1,1>>>(matrix,glcm,nx,Max);;
        hipDeviceSynchronize();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    else if(degree==315){
        blok1_315<<<1,1>>>(matrix,glcm,nx,Max);;
        hipDeviceSynchronize();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    end = clock();
    t = ((double) (end - start))/CLOCKS_PER_SEC;


    int sum;
    sum=0;
    for(int i=0;i<Max*Max;i++){
        sum +=transposed[i];
    }
    printf("sum %d",sum);
    normalization<<<Max,Max>>>(transposed,norm,Max,sum);


    hipDeviceSynchronize();
    printMatrixnxormalization(norm,Max,degree,nx);
    float sums;
    sums=0;
    for(int i=0;i<Max*Max;i++){
        sums  +=norm[i];
    }
    //Jumlah <<< Max,Max >>>(sumMatrix,norm);
    printf("jumlah %f\n",sums);
    dim3 b(32,32);
    dim3 g((Max + b.x - 1) / b.x, (Max + b.y - 1) / b.y);
    //Step1
    calculate_contrast<<<g,b>>>(norm,contrast,Max);
    hipDeviceSynchronize();
    calculate_entropy<<<g,b>>>(norm,entropy,Max);
    calculate_IDM<<<g,b>>>(norm,IDM,Max);
    calculate_ASM<<<g,b>>>(norm,ASM,Max);
    calculate_miu_i<<<g,b>>>(norm,miu_i,Max);
    hipDeviceSynchronize();
    calculate_miu_j<<<g,b>>>(norm,miu_j,Max);
    hipDeviceSynchronize();
    //Step2
    calculate_std_i<<<g,b>>>(norm,std_i,miu_i,Max);
    calculate_std_j<<<g,b>>>(norm,std_j,miu_j,Max);
    calculate_variance<<<g,b>>>(norm,variance,miu_i,Max);
    calculate_sumaverage<<<g,b>>>(norm,sav,Max);
    calculate_sumentropy<<<g,b>>>(norm,sen,Max);
    calculate_differenceentropy<<<g,b>>>(norm,den,Max);
    calculate_HX<<<g,b>>>(norm,HX,Max);
    calculate_HY<<<g,b>>>(norm,HY,Max);
    calculate_HXY1<<<g,b>>>(norm,HXY1,Max);
    hipDeviceSynchronize();
    //Step3
    std_j[0]=sqrt(std_j[0]);
    std_i[0]=sqrt(std_i[0]);
    calculate_sumvariance<<<g,b>>>(norm,sva,sen,Max);
    calculate_korelasi<<<g,b>>>(norm,korelasi,miu_i,std_i,miu_j,std_j,Max);
    calculate_dva<<<g,b>>>(norm,dva,Max);
    hipDeviceSynchronize();


    printf("ASM : %.3f\n",ASM[0]);
    printf("Contrast : %.3f\n",contrast[0]);
    printf("IDM : %.3f\n",IDM[0]);
    printf("entropy : %.7f\n",-(entropy[0]));
    printf("miu_i : %.3f\n",(miu_i[0]));
    printf("miu_j : %.3f\n",(miu_j[0]));
    printf("std_i : %.3f\n",(std_i[0]));
    printf("std_j : %.3f\n",(std_j[0]));
    printf("variance : %.3f\n",(variance[0]));
    printf("SAV : %.3f\n",(sav[0]));
    printf("SEN : %.3f\n",-(sen[0]));
    printf("SVA : %.3f\n",(sva[0]));
    printf("DEN : %.3f\n",-(den[0]));
    printf("HX : %.3f\n",-(HX[0]));
    printf("HY : %.3f\n",-(HY[0]));
    printf("HXY1 : %.7f\n",-(HXY1[0]));
    printf("IMC : %.7f\n",(entropy[0]-HXY1[0])/max(-(HX[0]),-(HY[0])));
    printf("korelasi : %.3f\n",(korelasi[0]));
    printf("Differnece Variance : %.3f\n",(dva[0]));

    printf("matrix gambar disimpan di matrix_gambar.txt\n");
    printf("matrix glcm disimpan di matrix_glcm_host.txt\n");
    printf("matrix glcm normalisasi disimpan di matrix_normalisasi_host.txt\n");


    printf("waktu eksekusi: %f\n",t);
    // free host and devide memory
    hipFree(matrix);hipFree(glcm);hipFree(norm);
    hipFree(mulMatrix);
}