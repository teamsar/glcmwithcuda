#include "hip/hip_runtime.h"

   #include <hip/hip_runtime.h>
   #include <hip/hip_runtime.h>
   #include <stdio.h>
   #include <stdlib.h>
   #include <time.h>
   #include "kuda.h"
   #include "lodepng.h"
// Banyak nx * nx Matrix
// Banyak Max * Max Matrix
int Max;

  
void printMatrixGambar(int *C, const int nx, const int ny)
{
    int *ic = C;
    //printf("\nMatrix Gambar: (%d.%d)\n", nx, ny);
    FILE * fp = NULL;
    fp = fopen("matrixgambar.txt", "w+");

    if(fp == NULL){
        printf("Error creating results file\n");
        exit(1);
    }
    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            
            fprintf(fp, " %d   ", ic[ix]);
   
        }
        fprintf(fp, "\n");
        ic += nx;
    }
    
    printf("\n");
    fclose(fp);
    return;
}


void printMatrixGlcm(int *C, const int Max,int degree,int gambar)
{
    int *ic = C;
    FILE * fp = NULL;
    if(gambar==128){
        if(degree==0){
            fp = fopen("./data/GLCM/Sample128/Sudut_0/matrix_glcm_0_method2.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/GLCM/Sample128/Sudut_90/matrix_glcm_90_method2.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/GLCM/Sample128/Sudut_180/matrix_glcm_180_method2.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/GLCM/Sample128/Sudut_270/matrix_glcm_270_method2.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/GLCM/Sample128/Sudut_45/matrix_glcm_45_method2.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/GLCM/Sample128/Sudut_135/matrix_glcm_135_method2.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/GLCM/Sample128/Sudut_225/matrix_glcm_225_method2.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/GLCM/Sample128/Sudut_315/matrix_glcm_315_method2.txt", "w+");
        }
    }
    else if(gambar==256){
        if(degree==0){
            fp = fopen("./data/GLCM/Sample256/Sudut_0/matrix_glcm_0_method2.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/GLCM/Sample256/Sudut_90/matrix_glcm_90_method2.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/GLCM/Sample256/Sudut_180/matrix_glcm_180_method2.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/GLCM/Sample256/Sudut_270/matrix_glcm_270_method2.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/GLCM/Sample256/Sudut_45/matrix_glcm_45_method2.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/GLCM/Sample256/Sudut_135/matrix_glcm_135_method2.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/GLCM/Sample256/Sudut_225/matrix_glcm_225_method2.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/GLCM/Sample256/Sudut_315/matrix_glcm_315_method2.txt", "w+");
        }
    }
    else if(gambar==512){
        if(degree==0){
            fp = fopen("./data/GLCM/Sample512/Sudut_0/matrix_glcm_0_method2.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/GLCM/Sample512/Sudut_90/matrix_glcm_90_method2.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/GLCM/Sample512/Sudut_180/matrix_glcm_180_method2.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/GLCM/Sample512/Sudut_270/matrix_glcm_270_method2.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/GLCM/Sample512/Sudut_45/matrix_glcm_45_method2.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/GLCM/Sample512/Sudut_135/matrix_glcm_135_method2.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/GLCM/Sample512/Sudut_225/matrix_glcm_225_method2.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/GLCM/Sample512/Sudut_315/matrix_glcm_315_method2.txt", "w+");
        }
    }
    else if(gambar==1024){
        if(degree==0){
            fp = fopen("./data/GLCM/Sample1024/Sudut_0/matrix_glcm_0_method2.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/GLCM/Sample1024/Sudut_90/matrix_glcm_90_method2.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/GLCM/Sample1024/Sudut_180/matrix_glcm_180_method2.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/GLCM/Sample1024/Sudut_270/matrix_glcm_270_method2.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/GLCM/Sample1024/Sudut_45/matrix_glcm_45_method2.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/GLCM/Sample1024/Sudut_135/matrix_glcm_135_method2.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/GLCM/Sample1024/Sudut_225/matrix_glcm_225_method2.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/GLCM/Sample1024/Sudut_315/matrix_glcm_315_method2.txt", "w+");
        }
    }
    

    if(fp == NULL){
        printf("Error creating results file\n");
        exit(1);
    }
    for (int iy = 0; iy <Max; iy++)
    {
        for (int ix = 0; ix <Max; ix++)
        {
            fprintf(fp, "%d  ", ic[ix]);

        }
        fprintf(fp, "\n\n");
        ic += (Max);

    }

    printf("\n");
    fclose(fp);
    return;
}

void printMatrixnxormalization(float *C, const int Max,int degree,int gambar)
{
    float *ic = C;
    FILE * fp = NULL;
    if(gambar==128){
        if(degree==0){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_0/matrix_normalisasi_0_method2.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_90/matrix_normalisasi_90_method2.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_180/matrix_normalisasi_180_method2.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_270/matrix_normalisasi_270_method2.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_45/matrix_normalisasi_45_method2.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_135/matrix_normalisasi_135_method2.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_225/matrix_normalisasi_225_method2.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/Normalisasi/Sample128/Sudut_315/matrix_normalisasi_315_method2.txt", "w+");
        }
    }
    else if(gambar==256){
        if(degree==0){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_0/matrix_normalisasi_0_method2.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_90/matrix_normalisasi_90_method2.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_180/matrix_normalisasi_180_method2.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_270/matrix_normalisasi_270_method2.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_45/matrix_normalisasi_45_method2.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_135/matrix_normalisasi_135_method2.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_225/matrix_normalisasi_225_method2.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/Normalisasi/Sample256/Sudut_315/matrix_normalisasi_315_method2.txt", "w+");
        }
    }
    else if(gambar==512){
        if(degree==0){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_0/matrix_normalisasi_0_method2.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_90/matrix_normalisasi_90_method2.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_180/matrix_normalisasi_180_method2.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_270/matrix_normalisasi_270_method2.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_45/matrix_normalisasi_45_method2.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_135/matrix_normalisasi_135_method2.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_225/matrix_normalisasi_225_method2.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/Normalisasi/Sample512/Sudut_315/matrix_normalisasi_315_method2.txt", "w+");
        }
    }
    else if(gambar==1024){
        if(degree==0){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_0/matrix_normalisasi_0_method2.txt", "w+");
        }
        else if(degree==90){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_90/matrix_normalisasi_90_method2.txt", "w+");
        }
        else if(degree==180){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_180/matrix_normalisasi_180_method2.txt", "w+");
        }
        else if(degree==270){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_270/matrix_normalisasi_270_method2.txt", "w+");
        }
        else if(degree==45){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_45/matrix_normalisasi_45_method2.txt", "w+");
        }
        else if(degree==135){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_135/matrix_normalisasi_135_method2.txt", "w+");
        }
        else if(degree==225){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_225/matrix_normalisasi_225_method2.txt", "w+");
        }
        else if(degree==315){
            fp = fopen("./data/Normalisasi/Sample1024/Sudut_315/matrix_normalisasi_315_method2.txt", "w+");
        }
    }
    if(fp == NULL){
        printf("Error creating results file\n");
        exit(1);
    }
    for (int iy = 0; iy < Max; iy++)
    {
        for (int ix = 0; ix <Max; ix++)
        {

            fprintf(fp, "%.7f  ", ic[ix]);

        }
        fprintf(fp, "\n\n");
        ic +=Max;

    }

    printf("\n");
    fclose(fp);
    return;
}



//calculate glcm
__global__ void Div0(int *matrix , int *newMatrix,int nx,int ny,int Max){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int Index = iy * nx + ix;
    int posisi = 0;

    for(int i = 0 ; i < nx ; i += 2){
        if(Index >= i * nx && Index < ((i + 1) * nx) - 1){

            posisi = matrix[Index] * Max + matrix[Index + 1];
            atomicAdd(&newMatrix[posisi],1);

            posisi = matrix[Index + nx] * Max + matrix[Index + (nx + 1)];
            atomicAdd(&newMatrix[posisi],1);
        }
    }
}

__global__ void Div45(int *matrix , int *newMatrix,int nx,int ny,int Max){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int Index = iy * nx + ix;
    int posisi = 0;

    for(int i = 0 ; i < nx - 1 ; i++){
        if(Index >= i * nx && Index < ((i + 1) * nx) - 1){
        posisi = matrix[Index + nx] * Max + matrix[Index + 1];
        atomicAdd(&newMatrix[posisi],1);
        }
    }
}

__global__ void Div90(int *matrix , int *newMatrix,int nx,int ny,int Max){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int Index = iy * nx + ix;
    int posisi = 0;

    for(int i = 0 ; i < nx - 1 ; ++i){
        if(Index >= i * nx && Index < ((i + 1) * nx) - 1){
            if(Index == 0 || Index % 2 == 0){
                posisi = matrix[Index + nx] * Max + matrix[Index];
                atomicAdd(&newMatrix[posisi],1);

                posisi = matrix[Index + (nx + 1)] * Max + matrix[Index + 1];
                atomicAdd(&newMatrix[posisi],1);
            }
        }
    }
}

__global__ void Div135(int *matrix , int *newMatrix,int nx,int ny,int Max){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int Index = iy * nx + ix;
    int posisi = 0;

    for(int i = 0 ; i < nx - 1 ; ++i){
        if(Index >= i * nx && Index < ((i + 1) * nx) - 1){

            posisi = matrix[Index + (nx + 1)] * Max + matrix[Index];
            atomicAdd(&newMatrix[posisi],1);
        }
    }
}

__global__ void Div180(int *matrix , int *newMatrix,int nx,int ny,int Max){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int Index = iy * nx + ix;
    int posisi = 0;

    for(int i = 0 ; i < nx ; i += 2){
        if(Index >= i * nx && Index < ((i + 1) * nx) - 1){

                posisi = matrix[Index + 1] * Max + matrix[Index];
                atomicAdd(&newMatrix[posisi],1);

                posisi = matrix[Index + (nx + 1)] * Max + matrix[Index + nx];
                atomicAdd(&newMatrix[posisi],1);
        }
    }
}

__global__ void Div225(int *matrix , int *newMatrix,int nx,int ny,int Max){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int Index = iy * nx + ix;
    int posisi = 0;

    for(int i = 0 ; i < nx - 1 ; ++i){
        if(Index >= i * nx && Index < ((i + 1) * nx) - 1){
            posisi = matrix[Index + 1] * Max + matrix[Index + nx];
            atomicAdd(&newMatrix[posisi],1);
        }
    }
}

__global__ void Div270(int *matrix , int *newMatrix,int nx,int ny,int Max){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int Index = iy * nx + ix;
    int posisi = 0;

    for(int i = 0 ; i < nx - 1 ; ++i){
        if(Index >= i * nx && Index < ((i + 1) * nx) - 1){
            if(Index == 0 || Index % 2 == 0){
                posisi = matrix[Index] * Max + matrix[Index + nx];
                atomicAdd(&newMatrix[posisi],1);

                posisi = matrix[Index + 1] * Max + matrix[Index + (nx + 1)];
                atomicAdd(&newMatrix[posisi],1);
            }
        }
    }
}

__global__ void Div315(int *matrix , int *newMatrix,int nx,int ny,int Max){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int Index = iy * nx + ix;
    int posisi = 0;

    for(int i = 0 ; i < nx - 1 ;  ++i ){
        if(Index >= i * nx && Index < ((i + 1) * nx) - 1){
            posisi = matrix[Index] * Max + matrix[Index + (nx + 1)];
            atomicAdd(&newMatrix[posisi],1);
        }
    }
}



__global__ void Mul(float *newMatrix,float *mulMatrix,int Max,float *sumMatrix){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    for (int k = 0; k < Max; k++) {
        // Accumulate results for a single element
        atomicAdd(&mulMatrix[row * Max + col],newMatrix[row * Max + k] * newMatrix[k * Max + col]);
    }
}


__global__ void Jumlah(float *sumMatrix,float *mulMatrix){
    int Index = blockIdx.x * blockDim.x + threadIdx.x;
    atomicAdd(&sumMatrix[0],mulMatrix[Index]);

}

__global__ void AddToitTranspose(int *transposed,int *glcm,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;                
    int row = blockIdx.y * blockDim.y + threadIdx.y;
   

    transposed[row*Max+col]=glcm[row*Max+col]+glcm[col*Max+row];
    
}

__global__ void normalization(int *glcm,float *norm,int Max,int sum){
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * Max + ix;
    __syncthreads();
    if(idx<(Max+1)*(Max+1)){
        norm[idx]=float(glcm[idx])/float(sum);
    }
}
__global__ void calculate_contrast(float *norm,float *contrast,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        atomicAdd(&contrast[0],((row-col)*(row-col))*norm[row*Max+col]);
    }
    
}

__global__ void calculate_IDM(float *norm,float *IDM,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(norm[row*Max+col]>0){
        atomicAdd(&IDM[0],norm[row*Max+col] / (1+((row-col)*(row-col))) );
    }

}

__global__ void calculate_entropy(float *norm,float *entropy,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(norm[row*Max+col]>0){
        atomicAdd(&entropy[0],(norm[row*Max+col] * log10f(norm[row*Max+col])) );

    }

}


__global__ void calculate_ASM(float *norm,float *ASM,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(norm[row*Max+col]>0){
        atomicAdd(&ASM[0],norm[row*Max+col]*norm[row*Max+col]);
    }
}

__global__ void calculate_miu_i(float *norm,float *miu_i,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(norm[row*Max+col]>0){
        atomicAdd(&miu_i[0],row*norm[row*Max+col]);
    }
}



__global__ void calculate_miu_j(float *norm,float *miu_j,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(norm[row*Max+col]>0){
        atomicAdd(&miu_j[0],col*norm[row*Max+col]);
    }
}

__global__ void calculate_std_i(float *norm,float *std_i,float*miu_i,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(norm[row*Max+col]>0){
        atomicAdd(&std_i[0],norm[row*Max+col] * ((row-miu_i[0])*(row-miu_i[0])));
    }
}


__global__ void calculate_std_j(float *norm,float *std_i,float *miu_j,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(norm[row*Max+col]>0){
        atomicAdd(&std_i[0],norm[row*Max+col]*(((col-miu_j[0])*(col-miu_j[0]))));
    }

}__global__ void calculate_korelasi(float *norm,float *korelasi,float *miu_i,float *std_i,float *miu_j,float *std_j,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(norm[row*Max+col]>0){
        atomicAdd(&korelasi[0],(((row-miu_i[0])*(col-miu_j[0]))*norm[row*Max+col])/(std_i[0]*std_j[0]));

    }
}

__global__ void calculate_variance(float *norm,float *variance,float *miu_i,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        atomicAdd(&variance[0],((row-miu_i[0])*(row-miu_i[0]))*norm[row*Max+col]);

    }
}

__global__ void calculate_sumaverage(float *norm,float *sav,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int k;
    for(k=2;k<2*Max;k++){
        if((row+col)==k){
            atomicAdd(&sav[0],k*(1*norm[row*Max+col]));
        }
        else{
            atomicAdd(&sav[0],0);
        }
    }
}

__global__ void calculate_sumentropy(float *norm,float *sen,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int k;
    for(k=2;k<2*Max;k++){
        if((row+col)==k && norm[row*Max+col]>0){
            atomicAdd(&sen[0],(1*norm[row*Max+col])*(log10(1*norm[row*Max+col])));
        }
        else{
            atomicAdd(&sen[0],0);
        }
    }
}

__global__ void calculate_sumvariance(float *norm,float *sva,float *sen,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int k;
    for(k=2;k<2*Max;k++){
        if((row+col)==k && norm[row*Max+col]>0){
            atomicAdd(&sva[0],((k-sen[0])*(k-sen[0]))*(1*norm[row*Max+col]));
        }
        else{
            atomicAdd(&sva[0],0);
        }
    }
}

__global__ void calculate_differenceentropy(float *norm,float *den,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int k;
    for(k=0;k<Max-1;k++){
        if(abs(row-col)==k && norm[row*Max+col]>0){
            atomicAdd(&den[0],(1*norm[row*Max+col])*(log10(1*norm[row*Max+col])));
        }
        else{
            atomicAdd(&den[0],0);
        }
    }
}

__global__ void calculate_HX(float *norm,float *HX,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        atomicAdd(&HX[0],norm[row*Max+col]*log10f(norm[row*Max+col]));
    }
}


__global__ void calculate_HY(float *norm,float *HY,int Max){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(norm[row*Max+col]>0){
        atomicAdd(&HY[0],norm[row*Max+col]*log10f(norm[row*Max+col]));
        
    }
}

__global__ void calculate_HXY1(float *norm,float *HXY1,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(norm[row*Max+col]>0){
        atomicAdd(&HXY1[0],norm[row*Max+col]*log10f(norm[row*Max+col]));
    }
}

__global__ void calculate_dva(float *norm,float *dva,int Max){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int k;
    for(k=0;k<Max-1;k++){
        if(abs(row-col)==k && norm[row*Max+col]>0){
            atomicAdd(&dva[0],(k*k)*(1*norm[row*Max+col]));
        }
        else{
            atomicAdd(&dva[0],0);
        }
    }
}

void takeimagevalue(const char* filename, rgb_image *img)
{

     unsigned error;
     unsigned char* png;
     size_t pngsize;;

     lodepng_load_file(&png, &pngsize, filename);
     error = lodepng_decode32(&img->image, &img->width, &img->height, png, pngsize);

     if(error) printf("error %u: %s\n", error, lodepng_error_text(error));

}

void transformToGrayCuda(rgb_image *img){
	unsigned char* image = img->image;
    unsigned char* image_d;
    unsigned int  width = img->width;
    unsigned int height = img->height;
    int n =width*height;
    size_t size = n * 4 * sizeof(unsigned char);


	int device_count = 0;
	hipError_t status = hipGetDeviceCount(&device_count);

	status = hipMalloc((void **) &image_d, size);


	hipMemcpy(image_d, image,  size, hipMemcpyHostToDevice);

	dim3 block_size(16, 16);
	dim3 num_blocks(img->width / block_size.x, img->height / block_size.y);
    setPixelToGrayscale<<<num_blocks, block_size>>>(image_d, img->width, img->height);



	hipMemcpy(image, image_d, size, hipMemcpyDeviceToHost);

	hipFree(image_d);
}


__global__
void setPixelToGrayscale(unsigned char *image, unsigned width, unsigned height)
{
    float gray;
    float r, g, b;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		r = image[4 * width * y + 4 * x + 0];
		g = image[4 * width * y + 4 * x + 1];
		b = image[4 * width * y + 4 * x + 2];
		gray =.299f*r + .587f*g + .114f*b;
		image[4 * width * y + 4 * x + 0] = gray;
		image[4 * width * y + 4 * x + 1] = gray;
		image[4 * width * y + 4 * x + 2] = gray;
		image[4 * width * y + 4 * x + 3] = 255;
	}

}

void saveimagegray(const char* filename, rgb_image *img)
{
  /*Encode the image*/
  unsigned error = lodepng_encode32_file(filename, img->image, img->width, img->height);

  /*if there's an error, display it*/
  if(error) printf("error %u: %s\n", error, lodepng_error_text(error));
}

int main(int argc, char *argv[]){


    char *d;
    long deg =strtol(argv[2],&d,10);
    int degree=deg;
    printf("%s %d degre Starting...\n", argv[0],degree);
    const char* filename = argc > 1 ? argv[1] : "test.png";
    rgb_image img;
    takeimagevalue(filename, &img);
    transformToGrayCuda(&img); 
    int nx =img.width;
    int ny =img.height;
    printf("%d %d\n",nx,ny);
    int *matrix,*glcm,*transposed;
    float *norm,*mulMatrix,*sumMatrix;
    float*ASM,*contrast,*IDM,*entropy,*miu_i,*miu_j,*std_i,*std_j,*korelasi,*variance,*sav,*sen,*sva,*den,*HX,*HY,*HXY1,*dva;
    hipMallocManaged(&matrix, (nx * ny) * sizeof(int));

    for(int i = 0 ; i < (nx * nx) ; ++i){
        matrix[i] = img.image[i];
        if(matrix[i] > Max){
            Max = matrix[i];
        }
    }
    printMatrixGambar(matrix,nx,ny);

    Max = Max + 1; // karena index dimulai dari 0 dan Maximum 3 ( 0 - 3 = 4 ) jadi Max ditambah 1;

    hipMallocManaged(&glcm, (Max * Max) * sizeof(int));
    hipMallocManaged(&transposed, (Max * Max) * sizeof(int));
    hipMallocManaged(&mulMatrix, (Max * Max) * sizeof(float));
    hipMallocManaged(&sumMatrix, (Max * Max) * sizeof(float));
    hipMallocManaged(&norm, (Max * Max) * sizeof(float));
    for(int i = 0 ; i < (Max * Max) ; ++i){
        glcm[i] = 0;
        transposed[i] = 0;
        mulMatrix[i] = 0;
    }

   
   
    hipMallocManaged(&ASM, (Max * Max) * sizeof(float));
    hipMallocManaged(&contrast, (Max * Max) * sizeof(float));
    hipMallocManaged(&IDM, (Max * Max) * sizeof(float));
    hipMallocManaged(&entropy, (Max * Max) * sizeof(float));
    hipMallocManaged(&miu_i, (Max * Max) * sizeof(float));
    hipMallocManaged(&miu_j, (Max * Max) * sizeof(float));
    hipMallocManaged(&std_i, (Max * Max) * sizeof(float));
    hipMallocManaged(&std_j, (Max * Max) * sizeof(float));
    hipMallocManaged(&korelasi, (Max * Max) * sizeof(float));
    hipMallocManaged(&variance, (Max * Max) * sizeof(float));
    hipMallocManaged(&sav, (Max * Max) * sizeof(float));
    hipMallocManaged(&sen, (Max * Max) * sizeof(float));
    hipMallocManaged(&sva, (Max * Max) * sizeof(float));
    hipMallocManaged(&den, (Max * Max) * sizeof(float));
    hipMallocManaged(&HX, (Max * Max) * sizeof(float));
    hipMallocManaged(&HY, (Max * Max) * sizeof(float));
    hipMallocManaged(&dva, (Max * Max) * sizeof(float));
    hipMallocManaged(&HXY1, (Max * Max) * sizeof(float));

    dim3 block(2 ,2);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    dim3 grids((Max + block.x - 1) / block.x, (Max + block.y - 1) / block.y);
    clock_t start, end;
    double t = 0;
    start = clock();
    // invoke kernel for calculation
    if(degree==0){
        Div0<<<grid,block>>>(matrix,glcm, nx, ny,Max);
        hipDeviceSynchronize();
        end = clock();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    else if(degree ==180){
        Div180<<<grid,block>>>(matrix,glcm, nx, ny,Max);
        hipDeviceSynchronize();
        end = clock();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    else if(degree==270){
        Div270<<<grid,block>>>(matrix,glcm, nx, ny,Max);
        hipDeviceSynchronize();
        end = clock();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    else if(degree==90){
        Div90<<<grid,block>>>(matrix,glcm, nx, ny,Max);
        hipDeviceSynchronize();
        end = clock();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    else if(degree==45){
        Div45<<<grid,block>>>(matrix,glcm, nx, ny,Max);
        hipDeviceSynchronize();
        end = clock();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    else if(degree==135){
        Div135<<<grid,block>>>(matrix,glcm, nx, ny,Max);
        hipDeviceSynchronize();
        end = clock();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    else if(degree==225){
        Div225<<<grid,block>>>(matrix,glcm, nx, ny,Max);
        hipDeviceSynchronize();
        end = clock();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    else if(degree==315){
        Div315<<<grid,block>>>(matrix,glcm, nx, ny,Max);
        hipDeviceSynchronize();
        end = clock();
        AddToitTranspose<<<grids,block>>>(transposed,glcm,Max);
        hipDeviceSynchronize();
        printMatrixGlcm(glcm,Max,degree,nx);
    }
    //hipDeviceSynchronize();

    
    t = ((double) (end - start))/CLOCKS_PER_SEC;

    int sum;
    sum=0;
    for(int i=0;i<Max*Max;i++){
        sum +=transposed[i];
    }
    printf("sum %d",sum);
    normalization<<<Max,Max>>>(transposed,norm,Max,sum);


    hipDeviceSynchronize();
    printMatrixnxormalization(norm,Max,degree,nx);
    float sums;
    sums=0;
    for(int i=0;i<Max*Max;i++){
        sums  +=norm[i];
    }
    //Jumlah <<< Max,Max >>>(sumMatrix,norm);
    printf("jumlah %f\n",sums);
    dim3 b(32,32);
    dim3 g((Max + b.x - 1) / b.x, (Max + b.y - 1) / b.y);
    //Step1
    calculate_contrast<<<g,b>>>(norm,contrast,Max);
    hipDeviceSynchronize();
    calculate_entropy<<<g,b>>>(norm,entropy,Max);
    calculate_IDM<<<g,b>>>(norm,IDM,Max);
    calculate_ASM<<<g,b>>>(norm,ASM,Max);
    calculate_miu_i<<<g,b>>>(norm,miu_i,Max);
    hipDeviceSynchronize();
    calculate_miu_j<<<g,b>>>(norm,miu_j,Max); 
    hipDeviceSynchronize();
    //Step2
    calculate_std_i<<<g,b>>>(norm,std_i,miu_i,Max);
    calculate_std_j<<<g,b>>>(norm,std_j,miu_j,Max);
    calculate_variance<<<g,b>>>(norm,variance,miu_i,Max);
    calculate_sumaverage<<<g,b>>>(norm,sav,Max);
    calculate_sumentropy<<<g,b>>>(norm,sen,Max);
    calculate_differenceentropy<<<g,b>>>(norm,den,Max);
    calculate_HX<<<g,b>>>(norm,HX,Max);
    calculate_HY<<<g,b>>>(norm,HY,Max);
    calculate_HXY1<<<g,b>>>(norm,HXY1,Max);
    hipDeviceSynchronize();
    //Step3
    std_j[0]=sqrt(std_j[0]);
    std_i[0]=sqrt(std_i[0]);
    calculate_sumvariance<<<g,b>>>(norm,sva,sen,Max);
    calculate_korelasi<<<g,b>>>(norm,korelasi,miu_i,std_i,miu_j,std_j,Max);
    calculate_dva<<<g,b>>>(norm,dva,Max);
    hipDeviceSynchronize();

    
    printf("ASM : %.3f\n",ASM[0]);
    printf("Contrast : %.3f\n",contrast[0]);
    printf("IDM : %.3f\n",IDM[0]);
    printf("entropy : %.7f\n",-(entropy[0]));
    printf("miu_i : %.3f\n",(miu_i[0]));
    printf("miu_j : %.3f\n",(miu_j[0]));
    printf("std_i : %.3f\n",(std_i[0]));
    printf("std_j : %.3f\n",(std_j[0]));
    printf("variance : %.3f\n",(variance[0]));
    printf("SAV : %.3f\n",(sav[0]));
    printf("SEN : %.3f\n",-(sen[0]));
    printf("SVA : %.3f\n",(sva[0]));
    printf("DEN : %.3f\n",-(den[0]));
    printf("HX : %.3f\n",-(HX[0]));
    printf("HY : %.3f\n",-(HY[0]));
    printf("HXY1 : %.7f\n",-(HXY1[0]));
    printf("IMC : %.7f\n",(entropy[0]-HXY1[0])/max(-(HX[0]),-(HY[0])));
    printf("korelasi : %.3f\n",(korelasi[0]));
    printf("Differnece Variance : %.3f\n",(dva[0]));

    printf("matrix gambar disimpan di matrixgambar.txt\n");
    printf("matrix glcm disimpan di matrix_glcm_%d.txt\n",degree);
    printf("matrix glcm normalisasi disimpan di matrix_ormalisasi_%d.txt\n",degree);


    printf("waktu eksekusi: %f\n",t);
    // free host and devide memory
    hipFree(matrix);hipFree(glcm);hipFree(norm);
    hipFree(mulMatrix);
}